/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#define __CUDA_ARCH__ 200


#include "tracer.h"
#include "volume.h"
#include "light.h"
#include "object.h"
#include "clippingobject.h"
#include "texture.h"
#include "bitmap.h"

DEVICE ExposureRender::Tracer*			gpTracer			= NULL;
DEVICE ExposureRender::Volume* 			gpVolumes			= NULL;
DEVICE ExposureRender::Light*			gpLights			= NULL;
DEVICE ExposureRender::Object*			gpObjects			= NULL;
DEVICE ExposureRender::ClippingObject*	gpClippingObjects	= NULL;
DEVICE ExposureRender::Texture*			gpTextures			= NULL;
DEVICE ExposureRender::Bitmap*			gpBitmaps			= NULL;

#include "list.cuh"

ExposureRender::Cuda::List<ExposureRender::Tracer, ExposureRender::ErTracer>					gTracers("gpTracer");
ExposureRender::Cuda::List<ExposureRender::Volume, ExposureRender::ErVolume>					gVolumes("gpVolumes");
ExposureRender::Cuda::List<ExposureRender::Light, ExposureRender::ErLight>						gLights("gpLights");
ExposureRender::Cuda::List<ExposureRender::Object, ExposureRender::ErObject>					gObjects("gpObjects");
ExposureRender::Cuda::List<ExposureRender::ClippingObject, ExposureRender::ErClippingObject>	gClippingObjects("gpClippingObjects");
ExposureRender::Cuda::List<ExposureRender::Texture, ExposureRender::ErTexture>					gTextures("gpTextures");
ExposureRender::Cuda::List<ExposureRender::Bitmap, ExposureRender::ErBitmap>					gBitmaps("gpBitmaps");

#include "singlescattering.cuh"
#include "filterframeestimate.cuh"
#include "estimate.cuh"
#include "tonemap.cuh"

namespace ExposureRender
{

EXPOSURE_RENDER_DLL void BindTracer(const ErTracer& Tracer, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gTracers.Bind(Tracer);
	else
		gTracers.Unbind(Tracer);
}

EXPOSURE_RENDER_DLL void BindVolume(const ErVolume& Volume, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");

	if (Bind)
		gVolumes.Bind(Volume);
	else
		gVolumes.Unbind(Volume);
}

EXPOSURE_RENDER_DLL void BindLight(const ErLight& Light, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gLights.Bind(Light);
	else
		gLights.Unbind(Light);
}

EXPOSURE_RENDER_DLL void BindObject(const ErObject& Object, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gObjects.Bind(Object);
	else
		gObjects.Unbind(Object);
}

EXPOSURE_RENDER_DLL void BindClippingObject(const ErClippingObject& ClippingObject, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gClippingObjects.Bind(ClippingObject);
	else
		gClippingObjects.Unbind(ClippingObject);
}

EXPOSURE_RENDER_DLL void BindTexture(const ErTexture& Texture, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gTextures.Bind(Texture);
	else
		gTextures.Unbind(Texture);
}

EXPOSURE_RENDER_DLL void BindBitmap(const ErBitmap& Bitmap, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gBitmaps.Bind(Bitmap);
	else
		gBitmaps.Unbind(Bitmap);
}

EXPOSURE_RENDER_DLL void RenderEstimate(int TracerID)
{
	gTracers.Synchronize(TracerID);

	SingleScattering(gTracers[TracerID]);
	FilterFrameEstimate(gTracers[TracerID]);
	ComputeEstimate(gTracers[TracerID]);
	ToneMap(gTracers[TracerID]);

	gTracers[TracerID].NoIterations++;
}

EXPOSURE_RENDER_DLL void GetEstimate(int TracerID, unsigned char* pData)
{
	FrameBuffer& FB = gTracers[TracerID].FrameBuffer;

	Cuda::MemCopyDeviceToHost(FB.DisplayEstimate.GetData(), (ColorRGBAuc*)pData, FB.DisplayEstimate.GetNoElements());
}

EXPOSURE_RENDER_DLL void GetAutoFocusDistance(int TracerID, int FilmU, int FilmV, float& AutoFocusDistance)
{
//	ComputeAutoFocusDistance(FilmU, FilmV, AutoFocusDistance);
}

EXPOSURE_RENDER_DLL void GetNoIterations(int TracerID, int& NoIterations)
{
//	NoIterations = gTracers[TracerID].NoIterations; 
}

}
